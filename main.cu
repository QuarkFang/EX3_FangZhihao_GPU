#include "hip/hip_runtime.h"
#include <stdio.h>
#include <Windows.h>
#include <thrust\host_vector.h>
#include <thrust\device_vector.h>
#include <thrust\remove.h>
#include <thrust\execution_policy.h>
#include <thrust\remove.h>
#include <thrust\copy.h>
#include <thrust\extrema.h>
#include <list>
#include <cstdio>
#include <algorithm>
#include <iostream>
#include <ctime>
extern "C" {
#include <math.h>
#include "floating_number_helper.h"
#include "input_output.h"
}
#define INIT_LINE 100000
#define ERROR 0.000001
#define _CRT_SECURE_NO_WARNINGS
#define M_PI 3.14159265358979323846

//
// this function is what you need to finish
// @Usage : to solve the problem
// @Input : input containing all data needed
// @Output: answer containing all necessary data
//  you can find the definition of the two structs above in
//      input_output.h
//

double rm_error(double a) {
	long long b = a * 1000000000000;
	return b / 1000000000000.0;
}

struct rotation {
	double a0;
	double b0;

	rotation(double x, double y) { a0 = x, b0 = y; }

	__host__ __device__
		thrust::tuple<double, double, double> operator()(thrust::tuple<double, double, double> v) {
		double a = thrust::get<0>(v);
		double b = thrust::get<1>(v);
		double c = thrust::get<2>(v);
		double theta = atan2(b0, a0);
		double alpha = atan2(b, a);
		theta = M_PI / 2 - theta;
		double new_a = cos(theta)*a - sin(theta)*b;
		double new_b = sin(theta)*a + cos(theta)*b;
		double new_c = c;
		return thrust::make_tuple(new_a, new_b, new_c);
	}
};

struct if_plus {
	__host__ __device__
		bool operator()(thrust::tuple<double, double, double> v) {
		double b = thrust::get<1>(v);
		return (b > 0);
	}
};

struct if_minus {
	__host__ __device__
		bool operator()(thrust::tuple<double, double, double> v) {
		double b = thrust::get<1>(v);
		return (b < 0);
	}
};

struct slope {
	__host__ __device__
		line operator()(line v) {
		v.slope_value = -v.param_a/v.param_b;;
		return v;
	}
};

struct height {
	double x;
	height(double a) { x = a; }
	__host__ __device__
		double operator()(line v) {
		double a = v.param_a;
		double b = v.param_b;
		double c = v.param_c;
		double y = (c - a*x) / b;
		return y;
	}
};

typedef struct boundary {
	double x;
	double y_plus;
	double y_minus;
	line line_plus;
	line line_minus;
}boundary;

answer * compute(inputs * input) {
	int num = input->number;
	line ** lines = input->lines;
	answer * ans = (answer *)malloc(sizeof(answer));
	double start, stop, time;


	//
	// @todo write your codes here!
	//

	thrust::host_vector<double> h_a(num);
	thrust::host_vector<double> h_b(num);
	thrust::host_vector<double> h_c(num);

	for (int i = 0; i < num; i++) {
		h_a[i] = lines[i]->param_a;
		h_b[i] = lines[i]->param_b;
		h_c[i] = lines[i]->param_c;
	}

	thrust::device_vector<double> a = h_a;
	thrust::device_vector<double> b = h_b;
	thrust::device_vector<double> c = h_c;

	auto begin = thrust::make_zip_iterator(thrust::make_tuple(a.begin(), b.begin(), c.begin()));
	auto end = thrust::make_zip_iterator(thrust::make_tuple(a.end(), b.end(), c.end()));

	double ao = input->obj_function_param_a;
	double bo = input->obj_function_param_b;

	rotation pred(ao, bo);
	thrust::transform(begin, end, begin, pred);

	thrust::device_vector<double> a_plus(num);
	thrust::device_vector<double> b_plus(num);
	thrust::device_vector<double> c_plus(num);
	thrust::device_vector<double> a_minus(num);
	thrust::device_vector<double> b_minus(num);
	thrust::device_vector<double> c_minus(num);

	auto plus_begin  = thrust::make_zip_iterator(thrust::make_tuple(a_plus.begin(), b_plus.begin(), c_plus.begin()));
	auto minus_begin = thrust::make_zip_iterator(thrust::make_tuple(a_minus.begin(), b_minus.begin(), c_minus.begin()));
	auto plus_end    = thrust::make_zip_iterator(thrust::make_tuple(a_plus.end(), b_plus.end(), c_plus.end()));
	auto minus_end   = thrust::make_zip_iterator(thrust::make_tuple(a_minus.end(), b_minus.end(), c_minus.end()));

	plus_end = thrust::copy_if(begin, end, plus_begin, if_plus());
	minus_end = thrust::copy_if(begin, end, minus_begin, if_minus());

	int plus_count = num - thrust::count(a_plus.begin(), a_plus.end(), 0);
	int minus_count = num - thrust::count(a_minus.begin(), a_minus.end(), 0);

	//���ﻹ���ͷ�cuda�ڴ�
	/*
	thrust::device_vector<double> a_Ip(plus_count);
	thrust::device_vector<double> b_Ip(plus_count);
	thrust::device_vector<double> c_Ip(plus_count);
	thrust::device_vector<double> a_Im(minus_count);
	thrust::device_vector<double> b_Im(minus_count);
	thrust::device_vector<double> c_Im(minus_count);
	*/
	
	a_plus.erase(a_plus.begin() + plus_count, a_plus.begin() + num);
	b_plus.erase(b_plus.begin() + plus_count, b_plus.begin() + num);
	c_plus.erase(c_plus.begin() + plus_count, c_plus.begin() + num);
	a_minus.erase(a_minus.begin() + minus_count, a_minus.begin() + num);
	b_minus.erase(b_minus.begin() + minus_count, b_minus.begin() + num);
	c_minus.erase(c_minus.begin() + minus_count, c_minus.begin() + num);


	thrust::host_vector<line> h_I_plus(plus_count);
	thrust::host_vector<line> h_I_minus(minus_count);

	start = std::clock();
	for (int i = 0; i < plus_count; i++) {
		h_I_plus[i].param_a = a_plus[i];
		h_I_plus[i].param_b = b_plus[i];
		h_I_plus[i].param_c = c_plus[i];
	}
	for (int i = 0; i < minus_count; i++) {
		h_I_minus[i].param_a = a_minus[i];
		h_I_minus[i].param_b = b_minus[i];
		h_I_minus[i].param_c = c_minus[i];
	}
	stop = std::clock();
	time = ((double)(stop - start)) / CLK_TCK;
	std::cout << time << std::endl;

	thrust::device_vector<line> I_plus = h_I_plus;
	thrust::device_vector<line> I_minus = h_I_minus;

	thrust::transform(I_plus.begin(), I_plus.end(), I_plus.begin(), slope());
	thrust::transform(I_minus.begin(), I_minus.end(), I_minus.begin(), slope());

	boundary left_line;
	boundary right_line;
	boundary test_line;

	left_line.x = -INIT_LINE;
	right_line.x = INIT_LINE;
	test_line.x = 0;

	thrust::device_vector<double> LI_plus_height(plus_count);
	thrust::device_vector<double> LI_minus_height(minus_count);
	thrust::device_vector<double> RI_plus_height(plus_count);
	thrust::device_vector<double> RI_minus_height(minus_count);
	thrust::device_vector<double> TI_plus_height(plus_count);
	thrust::device_vector<double> TI_minus_height(minus_count);

	thrust::transform(I_plus.begin(), I_plus.end(), LI_plus_height.begin(), height(left_line.x));
	thrust::transform(I_minus.begin(), I_minus.end(), LI_minus_height.begin(), height(left_line.x));
	thrust::transform(I_plus.begin(), I_plus.end(), RI_plus_height.begin(), height(right_line.x));
	thrust::transform(I_minus.begin(), I_minus.end(), RI_minus_height.begin(), height(right_line.x));
	thrust::transform(I_plus.begin(), I_plus.end(), TI_plus_height.begin(), height(test_line.x));
	thrust::transform(I_minus.begin(), I_minus.end(), TI_minus_height.begin(), height(test_line.x));
	// ������Ը���

	

	unsigned int position;
	thrust::device_vector<double>::iterator iter;
	iter = thrust::max_element(LI_plus_height.begin(), LI_plus_height.end());
	position = iter - LI_plus_height.begin();
	left_line.line_plus = I_plus[position],   left_line.y_plus=LI_plus_height[position];
	iter = thrust::min_element(LI_minus_height.begin(), LI_minus_height.end());
	position = iter - LI_minus_height.begin();
	left_line.line_minus = I_minus[position], left_line.y_minus = LI_minus_height[position];		//L
	iter = thrust::max_element(RI_plus_height.begin(), RI_plus_height.end());
	position = iter - RI_plus_height.begin();
	right_line.line_plus = I_plus[position],  right_line.y_plus = RI_plus_height[position];
	iter = thrust::min_element(RI_minus_height.begin(), RI_minus_height.end());
	position = iter - RI_minus_height.begin();
	right_line.line_minus = I_minus[position],right_line.y_minus = RI_minus_height[position];		//R
	iter = thrust::max_element(TI_plus_height.begin(), TI_plus_height.end());
	position = iter - TI_plus_height.begin();
	test_line.line_plus = I_plus[position],   test_line.y_plus = TI_plus_height[position];
	iter = thrust::min_element(TI_minus_height.begin(), TI_minus_height.end());
	position = iter - TI_minus_height.begin();
	test_line.line_minus = I_minus[position], test_line.y_minus = TI_minus_height[position];		//T

	bool if_get_ans = FALSE;
	double res_x, res_y;
	while (!if_get_ans) {
		double plus_slope = test_line.line_plus.slope_value;
		double minus_slope = test_line.line_minus.slope_value;

		if (rm_error(test_line.y_plus) >rm_error(test_line.y_minus)) {
			if (plus_slope > 0) {
				if (minus_slope > plus_slope)
					left_line = test_line;
				else
					right_line = test_line;

			}
			else if (plus_slope < 0) {
				if (minus_slope < plus_slope)
					right_line = test_line;
				else
					left_line = test_line;
			}
			test_line.x = generate_intersection_point(&test_line.line_minus, &test_line.line_plus)->pos_x;
			thrust::transform(I_plus.begin(), I_plus.end(), TI_plus_height.begin(), height(test_line.x));
			thrust::transform(I_minus.begin(), I_minus.end(), TI_minus_height.begin(), height(test_line.x));
			iter = thrust::max_element(TI_plus_height.begin(), TI_plus_height.end());
			position = iter - TI_plus_height.begin();
			test_line.line_plus = I_plus[position], test_line.y_plus = TI_plus_height[position];
			iter = thrust::min_element(TI_minus_height.begin(), TI_minus_height.end());
			position = iter - TI_minus_height.begin();
			test_line.line_minus = I_minus[position], test_line.y_minus = TI_minus_height[position];
		}
		else if(rm_error(test_line.y_plus) <=rm_error(test_line.y_minus)){
			if (plus_slope > 0) {
				double inter_x = generate_intersection_point(&test_line.line_minus, &test_line.line_plus)->pos_x;
				if (minus_slope > plus_slope && inter_x > right_line.x) {
					if (rm_error(test_line.y_plus) == rm_error(test_line.y_minus)) {
						res_x = test_line.x;
						res_y = test_line.y_plus;
						ans->line1 = &test_line.line_plus;
						ans->line2 = &test_line.line_minus;
						if_get_ans = TRUE;
					}
					test_line.x = generate_intersection_point(&test_line.line_minus, &test_line.line_plus)->pos_x;
					thrust::transform(I_plus.begin(), I_plus.end(), TI_plus_height.begin(), height(test_line.x));
					thrust::transform(I_minus.begin(), I_minus.end(), TI_minus_height.begin(), height(test_line.x));
					iter = thrust::max_element(TI_plus_height.begin(), TI_plus_height.end());
					position = iter - TI_plus_height.begin();
					test_line.line_plus = I_plus[position], test_line.y_plus = TI_plus_height[position];
					iter = thrust::min_element(TI_minus_height.begin(), TI_minus_height.end());
					position = iter - TI_minus_height.begin();
					test_line.line_minus = I_minus[position], test_line.y_minus = TI_minus_height[position];
				}
				else {
					right_line = test_line;
					double intersection_x = generate_intersection_point(&right_line.line_plus, &left_line.line_plus)->pos_x;
					if (intersection_x == NULL) {
						res_x = test_line.x;
						res_y = test_line.y_plus;
						ans->line1 = &left_line.line_plus;
						ans->line2 = &right_line.line_plus;
						if_get_ans = TRUE;
					}
					else if(rm_error(intersection_x)== rm_error(test_line.x)) {
						res_x = test_line.x;
						res_y = test_line.y_plus;
						ans->line1 = &left_line.line_plus;
						ans->line2 = &right_line.line_plus;
						if_get_ans = TRUE;
					}
					else {
						test_line.x = intersection_x;
						thrust::transform(I_plus.begin(), I_plus.end(), TI_plus_height.begin(), height(test_line.x));
						thrust::transform(I_minus.begin(), I_minus.end(), TI_minus_height.begin(), height(test_line.x));
						iter = thrust::max_element(TI_plus_height.begin(), TI_plus_height.end());
						position = iter - TI_plus_height.begin();
						test_line.line_plus = I_plus[position], test_line.y_plus = TI_plus_height[position];
						iter = thrust::min_element(TI_minus_height.begin(), TI_minus_height.end());
						position = iter - TI_minus_height.begin();
						test_line.line_minus = I_minus[position], test_line.y_minus = TI_minus_height[position];
					}
				}
			}
			else if (plus_slope < 0) {
				double inter_x = generate_intersection_point(&test_line.line_minus, &test_line.line_plus)->pos_x;
				if (minus_slope < plus_slope &&  inter_x < right_line.x) {
					if (rm_error(test_line.y_plus) == rm_error(test_line.y_minus)) {
						res_x = test_line.x;
						res_y = test_line.y_plus;
						ans->line1 = &test_line.line_plus;
						ans->line2 = &test_line.line_minus;
						if_get_ans = TRUE;
					}
					test_line.x = generate_intersection_point(&test_line.line_minus, &test_line.line_plus)->pos_x;
					thrust::transform(I_plus.begin(), I_plus.end(), TI_plus_height.begin(), height(test_line.x));
					thrust::transform(I_minus.begin(), I_minus.end(), TI_minus_height.begin(), height(test_line.x));
					iter = thrust::max_element(TI_plus_height.begin(), TI_plus_height.end());
					position = iter - TI_plus_height.begin();
					test_line.line_plus = I_plus[position], test_line.y_plus = TI_plus_height[position];
					iter = thrust::min_element(TI_minus_height.begin(), TI_minus_height.end());
					position = iter - TI_minus_height.begin();
					test_line.line_minus = I_minus[position], test_line.y_minus = TI_minus_height[position];
				}
				else {
					left_line = test_line;
					double intersection_x = generate_intersection_point(&right_line.line_plus, &left_line.line_plus)->pos_x;
					if (intersection_x == NULL) {
						res_x = test_line.x;
						res_y = test_line.y_plus;
						ans->line1 = &left_line.line_plus;
						ans->line2 = &right_line.line_plus;
						if_get_ans = TRUE;
					}
					else if (rm_error(intersection_x) == rm_error(test_line.x)) {
						res_x = test_line.x;
						res_y = test_line.y_plus;
						ans->line1 = &left_line.line_plus;
						ans->line2 = &right_line.line_plus;
						if_get_ans = TRUE;
					}
					else {
						test_line.x = intersection_x;
						thrust::transform(I_plus.begin(), I_plus.end(), TI_plus_height.begin(), height(test_line.x));
						thrust::transform(I_minus.begin(), I_minus.end(), TI_minus_height.begin(), height(test_line.x));
						iter = thrust::max_element(TI_plus_height.begin(), TI_plus_height.end());
						position = iter - TI_plus_height.begin();
						test_line.line_plus = I_plus[position], test_line.y_plus = TI_plus_height[position];
						iter = thrust::min_element(TI_minus_height.begin(), TI_minus_height.end());
						position = iter - TI_minus_height.begin();
						test_line.line_minus = I_minus[position], test_line.y_minus = TI_minus_height[position];
					}
				}
			}
		}
	}

	entry:
	double theta = atan2(bo, -ao);
	theta = M_PI / 2 - theta;
	double new_x = cos(theta)*res_x - sin(theta)*res_y;
	double new_y = sin(theta)*res_x + cos(theta)*res_y;
	double a1 = cos(theta)*ans->line1->param_a - sin(theta)*ans->line1->param_b;
	double b1 = sin(theta)*ans->line1->param_a + cos(theta)*ans->line1->param_b;
	double a2 = cos(theta)*ans->line2->param_a - sin(theta)*ans->line2->param_b;
	double b2 = sin(theta)*ans->line2->param_a + cos(theta)*ans->line2->param_b;
	double c1 = ans->line1->param_c;
	double c2 = ans->line2->param_c;
	ans->line1 = generate_line_from_abc(a1, b1, c1);
	ans->line2 = generate_line_from_abc(a2, b2, c2);
	ans->intersection_point = generate_point_from_xy(new_x, new_y);
	ans->answer_b = ans->intersection_point->pos_x*input->obj_function_param_a + ans->intersection_point->pos_y*input->obj_function_param_b;
	//���ﻹ���ͷ�cuda�ڴ�

	return ans;
}

int main() {
	// 1. get the input data
	inputs * input = read_from_file("../test_cases/100000_0.dat");
	// 2. get the answer
	answer * ans = compute(input);
	for (int i = 0; i < 10; i++) {
		ans = compute(input);
	}
	// 3. display result and free memory
	char * ans_string = generate_ans_string(ans);
	printf("%s", ans_string);
	getchar();

	free_inputs(&input);
	free_ans(&ans);
	free(ans_string);
	return 0;
}